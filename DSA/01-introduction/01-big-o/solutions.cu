#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>

__global__ void max_reduction(int *data, int *result, int n) {
    extern __shared__ int sdata[];  // Shared memory for partial results
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    if (i < n) {
        sdata[tid] = data[i];
    } else {
        sdata[tid] = INT_MIN;  // If out of bounds, set to the minimum value
    }
    __syncthreads();

    // Reduce in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // Write result for this block
    if (tid == 0) result[blockIdx.x] = sdata[0];
}

int find_max_cuda(int *h_data, int n) {
    int *d_data, *d_result, *h_result;
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    h_result = (int*) malloc(blocksPerGrid * sizeof(int));
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_result, blocksPerGrid * sizeof(int));

    hipMemcpy(d_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    max_reduction<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_data, d_result, n);

    // Copy result back to host
    hipMemcpy(h_result, d_result, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

    // Final reduction on the CPU
    int max_value = h_result[0];
    for (int i = 1; i < blocksPerGrid; i++) {
        if (h_result[i] > max_value) {
            max_value = h_result[i];
        }
    }

    hipFree(d_data);
    hipFree(d_result);
    free(h_result);

    return max_value;
}

int main(int argc, char *argv[]) {
    int size = 1000;  // Default size, can be modified based on input
    if (argc > 1) {
        sscanf(argv[1], "%d", &size);  // Get size from command-line argument
    }

    // Dynamically allocate memory for the array to handle large input sizes
    int *nums = (int *)malloc(size * sizeof(int));
    if (nums == NULL) {
        printf("Memory allocation failed!\n");
        return 1;
    }

    // Fill array with values 0, 1, 2, ..., size-1
    for (int i = 0; i < size; i++) {
        nums[i] = i;
    }

    // Find max using CUDA
    int max_value = find_max_cuda(nums, size);

    printf("Max Value: %d\n", max_value);

    // Free the allocated memory
    free(nums);

    return 0;
}
